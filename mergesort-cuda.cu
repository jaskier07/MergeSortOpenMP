#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <cstring>
#include <algorithm> 
#include <cmath>
#include <hip/hip_runtime.h>


#define DBG 1
#define SHOWTIME 0

#define NUMBERS_BIG 100000000 //2000000
#define NUMBERS_DBG 128

#define MAX_BIG 100000 //1000000
#define MAX_DBG 99

#define NUMBERS ((DBG == 1) ? NUMBERS_DBG : NUMBERS_BIG)
#define MAX_NUMBER ((DBG == 1) ? MAX_DBG : MAX_BIG)

/* cuda errors */
bool checkForError(const hipError_t cudaStatus, const char text[], int* dev_input, int* dev_tmp) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		if (dev_input != NULL) {
			hipFree(dev_input);
		}
		if (dev_tmp != NULL) {
			hipFree(dev_tmp);
		}
		return true;
	}
	return false;
}

bool checkForError(const hipError_t cudaStatus, const char text[]) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		return true;
	}
	return false;
}

/* info */
void printArray(int* A, int size) {
	printf("\n");
	for (int i = 0; i < size; i++) {
		printf("%d, ", A[i]);
	}
	printf("\n");
	fflush(stdout);
}

void printTime(time_t t1, time_t t2, const char* solutionType) {
	printf("\nTime in seconds (mergesort %s): %f", solutionType, difftime(t2, t1));
}

void checkIfCorrectlySorted(int* arr) {
	bool correct = true;
	for (int i = 0; i < NUMBERS - 1; i++) {
		if (arr[i] > arr[i + 1]) {
			printf("\n\n-----------ERROR!-----------%d\n\n ", i);
			correct = false;
			break;
		}
	}
	if (correct) {
		printf("\n----------- OK ------------");
	}
}

/* merge sort */
void fillArrayWithNumbers(int* numbers) {
	int i;
	srand(time(NULL));
	for (i = 0; i < NUMBERS; i++) {
		numbers[i] = rand() % MAX_NUMBER;
	}

	if (DBG) {
		printArray(numbers, NUMBERS);
	}
}

__host__
__device__
int getMid(int start, int end) {
	return start + (end - start) / 2;
}

__host__
__device__
void merge(int* arr, int* tmp, int leftStart, int rightEnd, int mid) {
	int i, j, k;
	int leftHalfSize = mid - leftStart + 1;
	int rightHalfSize = rightEnd - mid;

	/* create temp arrays */
	int* L = &tmp[leftStart];
	int* R = &tmp[mid + 1];

	/* Copy data to temp arrays L[] and R[] */
	for (i = 0; i < leftHalfSize; i++) {
		L[i] = arr[leftStart + i];
	}
	for (j = 0; j < rightHalfSize; j++) {
		R[j] = arr[mid + 1 + j];
	}

	/* Merge the temp arrays back into arr[l..r]*/
	i = 0;
	j = 0;
	k = leftStart;
	while (i < leftHalfSize && j < rightHalfSize) {
		if (L[i] <= R[j]) {
			arr[k] = L[i];
			i++;
		}
		else {
			arr[k] = R[j];
			j++;
		}
		k++;
	}

	/* Copy the remaining elements of L[], if there are any */
	while (i < leftHalfSize) {
		arr[k] = L[i];
		i++;
		k++;
	}

	/* Copy the remaining elements of R[], if there are any */
	while (j < rightHalfSize) {
		arr[k] = R[j];
		j++;
		k++;
	}	
}

/* Function to merge the two haves arr[l..m] and arr[m+1..r] of array arr[] */
__global__
void mergeKernel(int* arr, int* tmp, int vectorLengthPerThread, int vectorLength) {
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	int leftStart = threadId * vectorLengthPerThread;
	int rightEnd = leftStart + vectorLengthPerThread - 1;
	int mid = getMid(leftStart, rightEnd);
	
	if (leftStart < vectorLength) {
		printf("\n thread: %d, <%d, %d>, mid %d", threadId, leftStart, rightEnd, mid);
		merge(arr, tmp, leftStart, rightEnd, mid);
	}
}


int main() {
	const int vectorLength = NUMBERS;
	const int threadsPerBlock = 64; // FIXME // 128
	int vectorLengthPerThread = 2; // FIXME
	const int vectorMultiplier = 2; // FIXME
	const int numBlocks = ceil(vectorLength / threadsPerBlock);
	const int blockVectorLength = vectorLength / numBlocks;
	const int vectorSizeInBytes = NUMBERS * sizeof(int);
	
	printf("\nConfiguration: vector length: %d, threads per block: %d, vector length per thread: %d, num blocks: %d",
		NUMBERS, threadsPerBlock, vectorLengthPerThread, numBlocks);

	int* tmp = (int*)malloc(vectorSizeInBytes);
	int* vector = (int*)malloc(vectorSizeInBytes);

	fillArrayWithNumbers(vector);

	int* dev_input = NULL;
	int* dev_tmp = NULL;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (checkForError(cudaStatus, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?", dev_input, dev_tmp)) {
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_input, vectorSizeInBytes);
	if (checkForError(cudaStatus, "hipMalloc (dev_input) failed!", dev_input, dev_tmp)) {
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_tmp, vectorSizeInBytes);
	if (checkForError(cudaStatus, "hipMalloc (dev_tmp) failed!", dev_input, dev_tmp)) {
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_input, vector, vectorSizeInBytes, hipMemcpyHostToDevice);
	if (checkForError(cudaStatus, "hipMemcpy (vector -> dev_input) failed!", dev_input, dev_tmp)) {
		return cudaStatus;
	}

	for (int i = 0; vectorLengthPerThread < blockVectorLength; i++,vectorLengthPerThread *= vectorMultiplier)  {		
		printf("\nIter: %d, vector length per thread: %d", i, vectorLengthPerThread);
		
		mergeKernel<<<numBlocks, threadsPerBlock>>>(dev_input, dev_tmp, vectorLengthPerThread, vectorLength);
		
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus, "mergeSortGlobal launch failed!", dev_input, dev_tmp)) {
			return cudaStatus;
		}		
		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus, "hipDeviceSynchronize on \"mergeSortGlobal\" returned error code.", dev_input, dev_tmp)) {
			return cudaStatus;
		}
		
		if (DBG) {
			cudaStatus = hipMemcpy(vector, dev_input, vectorSizeInBytes, hipMemcpyDeviceToHost);
			if (checkForError(cudaStatus, "hipMemcpy (dev_input -> vector) failed!")) {
				return cudaStatus;
			}
			printArray(vector, vectorLength);
		}
	}

	if (!DBG) {
		cudaStatus = hipMemcpy(vector, dev_input, vectorSizeInBytes, hipMemcpyDeviceToHost);
		if (checkForError(cudaStatus, "hipMemcpy (dev_input -> vector) failed!")) {
			return cudaStatus;
		}
	}
	
	merge(vector, tmp, 0, NUMBERS - 1, getMid(0, NUMBERS - 1));

	hipFree(dev_input);
	hipFree(dev_tmp);
	cudaStatus = hipDeviceReset();
	if (checkForError(cudaStatus, "hipDeviceReset failed!")) {
		return 1;
	}

	if (DBG) {
		printArray(vector, NUMBERS);
	}
	fflush(stdout);
	checkIfCorrectlySorted(vector);

	return 0;
}